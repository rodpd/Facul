
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecadd_cuda(double* a, double* b, double* c, int width) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;

  if (i < width)
    c[i] = a[i] + b[i];
}

void mm(double* a, double* b, double* c, int width) 
{
    int n = width*width;
    #pragma omp target map(tofrom: a[:n], b[:n], c[:n])
    #pragma omp teams distribute parallel for simd
    // #pragma omp parallel for private (i, j, k) schedule(static)
    for (int i = 0; i < width; i++) {
      for (int j = 0; j < width; j++) {
        double sum = 0;
        for (int k = 0; k < width; k++) {
          double x = a[i * width + k];
          double y = b[k * width + j];
          sum += x * y;
        }
        c[i * width + j] = sum;
      }
    }
}

int main()
{
  int width = 10000000;
  double *a = (double*) malloc (width * sizeof(double));
  double *b = (double*) malloc (width * sizeof(double));
  double *c = (double*) malloc (width * sizeof(double));

  for(int i = 0; i < width; i++) {
    a[i] = i;
    b[i] = width-i;
    c[i] = 0;
  }

  int size = width*sizeof(double);
  double *d_a, *d_b, *d_c;

  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_b, size);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_c, size);

  int block_size = 1024;
  dim3 dimGrid((width-1)/block_size + 1, 1, 1);
  dim3 dimBlock(block_size,1,1);

  vecadd_cuda<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < width; i++)
    printf("\n c[%d] = %f",i,c[i]);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

}
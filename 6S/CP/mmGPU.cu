/*
    TAMANHO = 500

    SEQUENCIAL:
    3.690s

    PARALELO MULTICORE:
    0.538s

    PARALELO GPU OPENMP:
    0.311s
    warps_launched = 2592
    warp_execution_efficiency = 21.73%

    PARALELO GPU CUDA:
    0.205s
    resultado do nvprof no parcode:
    ==2574== NVPROF is profiling process 2574, command: ./mmGPU
    ==2574== Profiling application: ./mmGPU
    ==2574== Profiling result:
    No events/metrics were profiled.
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void mm_cuda(double* a, double* b, double* c, int width) 
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  int j = blockIdx.y*blockDim.y+threadIdx.y;
    if ( i < width && j < width ) {
    double sum = 0;
    for ( int k = 0; k < width; k++ ) {
      double x = a[i*width+k];
      double y = b[k*width+j];
      sum += x*y;
    }
    c[i*width+j] = sum;
  }
}
 

void mm(double* a, double* b, double* c, int width) 
{
    int n = width*width;
    #pragma omp target map(tofrom: a[:n], b[:n], c[:n])
    #pragma omp teams distribute parallel for simd
    // #pragma omp parallel for schedule(static)
    for (int i = 0; i < width; i++) {
      for (int j = 0; j < width; j++) {
        double sum = 0;
        for (int k = 0; k < width; k++) {
          double x = a[i * width + k];
          double y = b[k * width + j];
          sum += x * y;
        }
        c[i * width + j] = sum;
      }
    }
}

int main()
{
  int width = 500;
  int n = width*width;
  double *a = (double*) malloc (width * width * sizeof(double));
  double *b = (double*) malloc (width * width * sizeof(double));
  double *c = (double*) malloc (width * width * sizeof(double));

    for(int i = 0; i < width; i++) {
        for(int j = 0; j < width; j++) {
            a[i*width+j] = i;
            b[i*width+j] = j;
            c[i*width+j] = 0;
        }
    }

  // mm(a,b,c,width);
 
  int tamanho = n*(sizeof(double));
  double *d_a, *d_b, *d_c;


  hipMalloc( &d_a, tamanho);
  hipMemcpy(d_a, a, tamanho, hipMemcpyHostToDevice);

  hipMalloc( &d_b, tamanho);
  hipMemcpy(d_b, b, tamanho, hipMemcpyHostToDevice);

  hipMalloc( &d_c, tamanho);

  int block_size = 2048;
  dim3 dimGrid((int)ceil(sqrt(n/block_size)),(int)ceil(sqrt(n/block_size)), 1);
  dim3 dimBlock((int)ceil(sqrt(block_size)),(int)ceil(sqrt(block_size)), 1);
  mm_cuda<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

  hipMemcpy(c, d_c, tamanho, hipMemcpyDeviceToHost);
 
/*
    for(int i = 0; i < width; i++) {
    for(int j = 0; j < width; j++) {
      printf("\n c[%d][%d] = %f",i,j,c[i*width+j]);
    }
   }
*/


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

}
